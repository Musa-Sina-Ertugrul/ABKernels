#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include "func_defs.h"
#include "definations.h"
#include "constants.h"
#include "gelu.h"

CREATE_FUNC_INTERMEADIATE(gelu_bfx2,        
        nv_bfloat162* local_input = BFX2_ptr(input);
        local_input[index] = __gelu_bfx2(local_input[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE(gelu_fx2,
        half2* local_input = FX2_ptr(input);
        local_input[index] = __gelu_fx2(local_input[index]);,half)

CREATE_FUNC_INTERMEADIATE(gelu_backward_bfx2,        
        nv_bfloat162* local_input = BFX2_ptr(input);
        local_input[index] = __gelu_backward_bfx2(local_input[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE(gelu_backward_fx2,
        half2* local_input = FX2_ptr(input);
        local_input[index] = __gelu_backward_fx2(local_input[index]);,half)

CREATE_CALL_FUNCTION_X2(gelu)

CREATE_CALL_FUNCTION_X2(gelu_backward)