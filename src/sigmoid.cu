#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include "func_defs.h"
#include "definations.h"
#include "sigmoid.h"

CREATE_FUNC_INTERMEADIATE(sigmoid_bfx2,        
        nv_bfloat162* local_input = BFX2_ptr(input);
        local_input[index] = __sigmoid_bfx2(local_input[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE(sigmoid_fx2,
        half2* local_input = FX2_ptr(input);
        local_input[index] = __sigmoid_fx2(local_input[index]);,half)

CREATE_FUNC_INTERMEADIATE(sigmoid_backward_bfx2,        
        nv_bfloat162* local_input = BFX2_ptr(input);
        local_input[index] = __sigmoid_backward_bfx2(local_input[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE(sigmoid_backward_fx2,
        half2* local_input = FX2_ptr(input);
        local_input[index] = __sigmoid_backward_fx2(local_input[index]);,half)

CREATE_CALL_FUNCTION_X2(sigmoid)

CREATE_CALL_FUNCTION_X2(sigmoid_backward)