#include "hip/hip_runtime.h"
#ifndef __FUNC_DEF__
#define __FUNC_DEF__

#include "./definations.cu"

#define CREATE_CALL_FUNCTION_X2_2INPUT(func_name) \
torch::Tensor \
func_name(torch::Tensor a,torch::Tensor b){\
    TORCH_CHECK(a.numel() == b.numel(),"Input arrays have different length");\
    TORCH_CHECK(a.device().index()==b.device().index(),"Input arrays have different devices");\
    TORCH_CHECK(a.dtype()==b.dtype(),"Input arrays have different dtypes");\
    TORCH_CHECK(a.numel() % 2 == 0, std::string("There is no such option for ") + STRINGIFY(func_name) + " two array use torch." + STRINGIFY(func_name));\
    torch::Tensor output = torch::zeros_like(a);\
    int device = a.device().index();\
    int block_count = (a.numel() / (2*THREAD_COUNT)) + 1;\
    hipStream_t stream = c10::cuda::getCurrentCUDAStream(device).stream();\
    switch (a.scalar_type())\
    {\
    case torch::kBFloat16:\
        CONCAT(func_name,_bfx2)<<<block_count,THREAD_COUNT, 0, stream>>>(BF_ptr(a.data_ptr()),BF_ptr(b.data_ptr()),BF_ptr(output.data_ptr()),a.numel()/2);\
        break;\
    case torch::kFloat16:\
        CONCAT(func_name,_fx2)<<<block_count,THREAD_COUNT, 0, stream>>>(F_ptr(a.data_ptr()),F_ptr(b.data_ptr()),F_ptr(output.data_ptr()),a.numel()/2);\
        break;\
    default:\
        TORCH_CHECK(false, std::string("There is no such option for ") + STRINGIFY(func_name) + " two array use torch." + STRINGIFY(func_name));\
        break;\
    }\
    hipStreamSynchronize(stream);\
    return output;\
}


#define CREATE_CALL_FUNCTION_X2(func_name) \
torch::Tensor \
func_name(torch::Tensor input){\
    TORCH_CHECK(input.numel() % 2 == 0, std::string("There is no such option for ") + STRINGIFY(gelu) + " two array use torch." + STRINGIFY(gelu));\
    int device = input.device().index();\
    int block_count = (input.numel() / (2*THREAD_COUNT)) + 1;\
    hipStream_t stream = c10::cuda::getCurrentCUDAStream(device).stream();\
    switch (input.scalar_type())\
    {\
    case torch::kBFloat16:\
        CONCAT(func_name,_bfx2)<<<block_count,THREAD_COUNT, 0, stream>>>(BF_ptr(input.data_ptr()),input.numel()/2);\
        break;\
    case torch::kFloat16:\
        CONCAT(func_name,_fx2)<<<block_count,THREAD_COUNT, 0, stream>>>(F_ptr(input.data_ptr()),input.numel()/2);\
        break;\
    default:\
        TORCH_CHECK(false, std::string("There is no such option for ") + STRINGIFY(func_name) + " two array use torch." + STRINGIFY(func_name));\
        break;\
    }\
    hipStreamSynchronize(stream);\
    return input;\
}

#define CREATE_ACTIVATION_FUNC_INNER(func_name,pre_calculation,calculation,dtype) \
static \
__device__ \
__forceinline__ \
dtype \
func_name(dtype input) { \
    do\
    {\
        pre_calculation\
        return calculation;\
    } while (0);\
}

#define CREATE_FUNC_INTERMEADIATE_2INPUT(func_name,inside,dtype) \
__global__ \
void \
func_name(dtype* array_left, dtype* array_right,dtype* output,uint64_t len){ \
    uint64_t index = blockIdx.x * blockDim.x + threadIdx.x; \
    if (index < len){ \
        inside \
    }\
}

#define CREATE_FUNC_INTERMEADIATE(func_name,inside,dtype) \
__global__ \
void \
func_name(dtype* input,u_int64_t len){ \
    uint64_t index = blockIdx.x * blockDim.x + threadIdx.x; \
    if (index< len){ \
        inside \
    } \
}


#endif