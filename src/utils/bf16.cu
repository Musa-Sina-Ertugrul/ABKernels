#include "hip/hip_runtime.h"
#ifndef __BF16__
#define __BF16__

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include "./func_defs.cu"

CREATE_ACTIVATION_FUNC_INNER(bf16_tanh,
    float2 input_fp32 = __bfloat1622float2(input);
    input_fp32.x = tanhf(input_fp32.x);
    input_fp32.y = tanhf(input_fp32.y);
    ,__float22bfloat162_rn(input_fp32),nv_bfloat162
)

#endif