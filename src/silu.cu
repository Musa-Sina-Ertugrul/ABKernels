#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include "func_defs.h"
#include "definations.h"
#include "sigmoid.h"
#include "silu.h"

CREATE_FUNC_INTERMEADIATE(silu_bfx2,        
        nv_bfloat162* local_input = BFX2_ptr(input);
        local_input[index] = __silu_bfx2(local_input[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE(silu_fx2,
        half2* local_input = FX2_ptr(input);
        local_input[index] = __silu_fx2(local_input[index]);,half)

CREATE_FUNC_INTERMEADIATE(silu_backward_bfx2,        
        nv_bfloat162* local_input = BFX2_ptr(input);
        local_input[index] = __silu_backward_bfx2(local_input[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE(silu_backward_fx2,
        half2* local_input = FX2_ptr(input);
        local_input[index] = __silu_backward_fx2(local_input[index]);,half)

CREATE_CALL_FUNCTION_X2(silu)

CREATE_CALL_FUNCTION_X2(silu_backward)