#include "hip/hip_runtime.h"
#ifndef __ADD__
#define __ADD__

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAStream.h>
#include "../utils/func_defs.cu"
#include "../utils/definations.cu"

CREATE_FUNC_INTERMEADIATE_2INPUT(add_bfx2,
        nv_bfloat162* local_left = BFX2_ptr(array_left);
        nv_bfloat162* local_right = BFX2_ptr(array_right);
        BFX2_ptr(output)[index] = __hadd2(local_left[index],local_right[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE_2INPUT(add_fx2,
        half2* local_left = FX2_ptr(array_left);
        half2* local_right = FX2_ptr(array_right);
        FX2_ptr(output)[index] = __hadd2(local_left[index],local_right[index]);, half)

CREATE_CALL_FUNCTION_X2_2INPUT(add)

#endif
