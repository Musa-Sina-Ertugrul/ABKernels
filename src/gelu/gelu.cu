#include "hip/hip_runtime.h"
#ifndef __GELU__
#define __GELU__

#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include "../utils/func_defs.cu"
#include "../utils/definations.cu"
#include "../utils/bf16.cu"

CREATE_ACTIVATION_FUNC_INNER(__gelu_bfx2,,__hmul2(
        __hmul2(__float2bfloat162_rn(0.5f),input),
        __hadd2(__float2bfloat162_rn(1.0f),
            h2tanh(
                __hmul2(__float2bfloat162_rn(0.45f),
                    __hadd2(input,
                        __hmul2(__float2bfloat162_rn(0.044715f),
                            __hmul2(__hmul2(input, input), input))))))),nv_bfloat162)

CREATE_ACTIVATION_FUNC_INNER(__gelu_fx2,, __hmul2(
        __hmul2(__float2half2_rn(0.5f),input),
        __hadd2(__float2half2_rn(1.0f),
            h2tanh(
                __hmul2(__float2half2_rn(0.45f),
                    __hadd2(input,
                        __hmul2(__float2half2_rn(0.044715f),
                            __hmul2(__hmul2(input, input), input))))))),half2)

// GELU backward for bfloat162
// Formula: grad_out * 0.5 * [(1 + tanh(z)) + x * c1 * sech²(z) * (1 + 3*c2*x²)]
// where z = c1*(x + c2*x³), c1=0.45, c2=0.044715
CREATE_ACTIVATION_FUNC_INNER(__gelu_backward_bfx2,// Compute x² and x³
    nv_bfloat162 x2 = __hmul2(input, input);
    nv_bfloat162 x3 = __hmul2(x2, input);

    // Compute z = c1*(x + c2*x³)
    nv_bfloat162 inner = __hadd2(input,
                                  __hmul2(__float2bfloat162_rn(0.044715f), x3));
    nv_bfloat162 z = __hmul2(__float2bfloat162_rn(0.45f), inner);

    // Compute tanh(z)
    nv_bfloat162 tanh_z = bf16_tanh(z);

    // Compute sech²(z) = 1 - tanh²(z)
    nv_bfloat162 tanh2_z = __hmul2(tanh_z, tanh_z);
    nv_bfloat162 sech2_z = __hsub2(__float2bfloat162_rn(1.0f), tanh2_z);

    // Compute (1 + 3*c2*x²)
    nv_bfloat162 term_x2 = __hadd2(__float2bfloat162_rn(1.0f),
                                    __hmul2(__float2bfloat162_rn(3.0f * 0.044715f), x2));

    // Compute x * c1 * sech²(z) * (1 + 3*c2*x²)
    nv_bfloat162 term2 = __hmul2(__hmul2(__hmul2(input, __float2bfloat162_rn(0.45f)),
                                          sech2_z),
                                  term_x2);

    // Compute (1 + tanh(z)) + term2
    nv_bfloat162 sum_terms = __hadd2(__hadd2(__float2bfloat162_rn(1.0f), tanh_z),
                                      term2);

    // Multiply by 0.5
    nv_bfloat162 grad_gelu = __hmul2(__float2bfloat162_rn(0.5f), sum_terms);,grad_gelu, nv_bfloat162)


// GELU backward for half2 (can use built-in h2tanh!)
// Formula: grad_out * 0.5 * [(1 + tanh(z)) + x * c1 * sech²(z) * (1 + 3*c2*x²)]
CREATE_ACTIVATION_FUNC_INNER(__gelu_backward_fx2,// Compute x² and x³
    half2 x2 = __hmul2(input, input);
    half2 x3 = __hmul2(x2, input);

    // Compute z = c1*(x + c2*x³)
    half2 inner = __hadd2(input,
                           __hmul2(__float2half2_rn(0.044715f), x3));
    half2 z = __hmul2(__float2half2_rn(0.45f), inner);

    // Compute tanh(z) - half2 HAS built-in tanh!
    half2 tanh_z = h2tanh(z);

    // Compute sech²(z) = 1 - tanh²(z)
    half2 tanh2_z = __hmul2(tanh_z, tanh_z);
    half2 sech2_z = __hsub2(__float2half2_rn(1.0f), tanh2_z);

    // Compute (1 + 3*c2*x²)
    half2 term_x2 = __hadd2(__float2half2_rn(1.0f),
                             __hmul2(__float2half2_rn(3.0f * 0.044715f), x2));

    // Compute x * c1 * sech²(z) * (1 + 3*c2*x²)
    half2 term2 = __hmul2(__hmul2(__hmul2(input, __float2half2_rn(0.45f)),
                                   sech2_z),
                           term_x2);

    // Compute (1 + tanh(z)) + term2
    half2 sum_terms = __hadd2(__hadd2(__float2half2_rn(1.0f), tanh_z),
                               term2);

    // Multiply by 0.5
    half2 grad_gelu = __hmul2(__float2half2_rn(0.5f), sum_terms);,grad_gelu,half2)

CREATE_FUNC_INTERMEADIATE(gelu_bfx2,        
        nv_bfloat162* local_input = BFX2_ptr(input);
        local_input[index] = __gelu_bfx2(local_input[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE(gelu_fx2,
        half2* local_input = FX2_ptr(input);
        local_input[index] = __gelu_fx2(local_input[index]);,half)

CREATE_FUNC_INTERMEADIATE(gelu_backward_bfx2,        
        nv_bfloat162* local_input = BFX2_ptr(input);
        local_input[index] = __gelu_backward_bfx2(local_input[index]);,hip_bfloat16)

CREATE_FUNC_INTERMEADIATE(gelu_backward_fx2,
        half2* local_input = FX2_ptr(input);
        local_input[index] = __gelu_backward_fx2(local_input[index]);,half)

CREATE_CALL_FUNCTION_X2(gelu)

CREATE_CALL_FUNCTION_X2(gelu_backward)

#endif